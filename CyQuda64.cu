#include "hip/hip_runtime.h"
#define WIN32_LEAN_AND_MEAN
#include <windows.h>
#include <stdio.h>
#include <assert.h>
#include <string>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <iostream>
#include <vector>

//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>

/*
 TO DO
  1. Search Entries using CUDA.
   2. Load entries from file
    3. Use a socket layer to accept commands.

  For "search entries", to copy the Array from Host to Device 
  we will need to flatten the array 
    from [Row][Column] 
	  to [Row + Column*N]
 
  Decide between Array, Vector, or Thrust
  where each has its drawbacks and limitations.

 */

using namespace std;

//Initialize Variables.
int matrixIndex = 0;    // Initial matrix Index.
int searchIndex = 0;    // Initial search Index.

int matrixSize = 8000;  // Max argvbase Entries.
int searchSize = 10;    // Max Search Entries.
int resultSize = 20;    // Max Result Entries.

int selectIndex = 0;
int i = 0;

// Create Arrays.
vector<vector<int>> myMatrix;
vector<vector<int>> mySearch;
vector<vector<int>> myResults;


hipError_t searchMatrix(int *searchResult, char *aMatrix, char *searchElement);
hipError_t addMatrix(char *matrixArray);

// CUDA engine for searchMatrix function.
__global__ void searchMatrixKernel(int searchResult, char aMatrix , char searchElement)
{
// int x = threadIdx.x;

/* 
 int sI = 0;
 // Detect the first matching character
 if (myMatrix[matrixIndex][sI] == searchElement[0]) {
   // Loop through next keyword character
   for (int j=1; j< matrixIndex.size(); j++) {
     if (myMatrix[matrixIndex][sI] != searchElement[j])
       break;
     else
     // Store the first matching character to the result list
       searchResult[sI] = 1;
   }
  }
*/

}

/*
// Search helper Function.
hipError_t searchMatrix(int * result, char *matrixargv, char *searchElements) 
{
	
 char *dev_argv = 0;
 char *dev_keyword = 0;
 int *dev_result = 0;
 
 hipError_t cudaStatus;
 cudaStatus = hipSetDevice(0);  // Choose which GPU to run on, change this on a multi-GPU system.

 if (cudaStatus != hipSuccess) { fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed? 0"); goto Error; }
 cudaStatus = hipMalloc((void**)&dev_result, resultSize * sizeof(int));                                          // Allocate GPU buffers for result set.
 if (cudaStatus != hipSuccess) {  fprintf(stderr, "hipMalloc failed! 1 "); goto Error;  }
 cudaStatus = hipMalloc((void**)&dev_argv, matrixSize * sizeof(char));                                           // Allocate GPU buffers for input argv set.
 if (cudaStatus != hipSuccess) {  fprintf(stderr, "hipMalloc failed! 2 ");  goto Error; }
 cudaStatus = hipMalloc((void**)&dev_keyword, sizeof(*searchElements));                                        // Allocate GPU buffers for keyword.
 if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed! 3 "); goto Error; }
 cudaStatus = hipMemcpy(dev_argv, matrixargv, matrixSize * sizeof(char), hipMemcpyHostToDevice);                      // Copy input argv from host memory to GPU buffers.
 if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed! 4 "); goto Error; }
 cudaStatus = hipMemcpy(dev_keyword, searchElements, sizeof(*searchElements), hipMemcpyHostToDevice);                // Copy keyword from host memory to GPU buffers.
 if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed! 5 "); goto Error; }

 searchMatrixKernel<<<1, matrixSize>>>(dev_result, dev_argv, dev_keyword);                                      // Launch a search keyword kernel on the GPU with one thread for each element.
 cudaStatus = hipDeviceSynchronize();                                                                         // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
 if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 6 \n", cudaStatus); goto Error; }
 cudaStatus = hipMemcpy(result, dev_result, resultSize * sizeof(int), hipMemcpyDeviceToHost);                  // Copy result from GPU buffer to host memory.
 if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed! 7"); goto Error; }

Error:
 hipFree(dev_result);
 hipFree(dev_argv);
 hipFree(dev_keyword);

 return cudaStatus;
}
*/

// Main Function.
int main(int argc, const char *argv[]) {
	
	std::string newArg;
    char *myOutput;
	myOutput = "";
	cout << "CyQuda-x64 1.0 (Console) \n\n";

getInput:
	newArg = "";
	cout << "> ";

	std::getline(std::cin, newArg);
	// Get Arguments.  std::vector<std::string> myArgs(argv, argv + argc);
	char split_char = ' ';
    std::istringstream split(newArg);
    std::vector<std::string> myCmd;
    for (std::string each; std::getline(split, each, split_char); myCmd.push_back(each));
    
	if (myCmd.size() == 0) { 
		cout << "usage:  CyQuda.exe Add/Request/Search/Exit \n\n";
		goto getInput;
	} 

	std::string my_Command = myCmd[0];

	if (my_Command == "Exit") { 
	    cout << "\n\n Exiting . . .\n\n";
		return 0;
	}
	
	if (myCmd.size() == 1) {
		if (my_Command == "Add") { cout << "\n\n Usage: Add 1.2.3 \n\n"; }
		if (my_Command == "Request") { cout << "\n\n Usage: Request 1 \n\n"; }
		if (my_Command == "Delete") { cout << "\n\n Usage: Delete 1 \n\n"; }
		if (my_Command == "Search") { cout << "\n\n Usage: Search 1.2.3 \n\n"; }
      goto getInput;
	}
	
    char split_char2 = '.';
    std::istringstream split2(myCmd[1]);
    std::vector<std::string> token;
    for (std::string each2; std::getline(split2, each2, split_char2); token.push_back(each2));
	
	// ================================== Add <Index Index ...>
	if (my_Command == "Add") {
	

     matrixIndex++;
	 if ((matrixIndex +1) > myMatrix.size()) { 
		 myMatrix.resize(myMatrix.size() + 1000);
	 }
	 myMatrix[matrixIndex].resize(token.size() + 1);
     // myMatrix.push_back(myRow);
     // myMatrix[matrixIndex].push_back(1);

	 cout << " - ";
	 for (i = 0; i < token.size(); i++) {
      myMatrix[matrixIndex][i] = atoi(token[i].c_str());
	 cout << token[i].c_str() << " ";
	 }
	 cout << "\n";
	 // sprintf(myOutput, "Index %d", matrixIndex);
	 
     goto getInput;
	}

	// ================================== Delete
	if (my_Command == "Delete") {
	 goto getInput;
	}

	// ================================== Request <Index>
	if (my_Command == "Request") {
        selectIndex = atoi(token[0].c_str());
		
		if (selectIndex > matrixIndex) {
			cout << "\n Out of Range \n"; 
			goto getInput;
		}
		if (selectIndex < 1) { 
			cout << "\n Out of Range \n"; 
			goto getInput;
		}
		cout << "\n Length: " << (myMatrix[selectIndex].size() -1) << "\n ";
	    cout << "\n ";
		for (i = 0; i < (myMatrix[selectIndex].size() -1); i++) {
   	     cout << myMatrix[selectIndex][i] << " "; 
		}

		cout << " \n";
	
		goto getInput;
	}
	
	// ================================== List	
	// = List active searches.
	if (my_Command == "List") {

	}

	// ================================== Search <element.element.element...> 
	// = Matches groups of elements from the token[x] array.  
    // =  returns "X <Search Index>" when completed.
	// =  Completed search indexes are re-used after completion.

    if (my_Command == "Search") {
	 if (matrixIndex == 0) { 
		 cout << "- No entries added. \n";
		 goto getInput;
	 }
	 
	 searchIndex++;
     mySearch[searchIndex].resize(token.size() + 1);
    
	 for (i = 0; i < token.size(); i++) {
      mySearch[searchIndex][i] = atoi(token[i].c_str());
	 }

   // Search Matrix (Not yet fully implimented.)
   //  hipError_t cudaStatus = searchMatrix(myResults, myMatrix, mySearch);
   //  if (cudaStatus != hipSuccess) { cout << "searchMatrix() failed! /n";  }

	 goto getInput;
	}

    // Load matrix array into CUDA memory.
	if (my_Command == "Matrix") { 

	}

	cout << "\n> ";
	goto getInput;

}
