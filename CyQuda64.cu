#define WIN32_LEAN_AND_MEAN
#include <stdio.h>
#include <assert.h>
#include <string>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <iostream>
#include <vector>

#include <winsock2.h>
#include <ws2tcpip.h>
#include <windows.h>

#pragma comment (lib, "Ws2_32.lib")
#include "CyQu_bridge.h"

#define DEFAULT_BUFLEN 512
#define DEFAULT_PORT "12008"

using namespace std;

void CySend(SOCKET ClientSocket, string cyData) {
	cyData = cyData + "\n";
	char *sendData;
	sendData = new char[cyData.size() + 1];
    memcpy(sendData, cyData.c_str(), cyData.size() + 1);

    int iSendResult; 
	iSendResult = send(ClientSocket,  sendData, cyData.size(), 0);
			
	        // iSendResult = send( ClientSocket, recvbuf, iResult, 0 );
            if (iSendResult == SOCKET_ERROR) {
                cout << "Send Failure: " << WSAGetLastError();
                closesocket(ClientSocket);
                WSACleanup();
                return;
            }
 return;			
}

int __cdecl init_Server(void) 
{
	
	int CQ_success;
    int iResult;

	WSADATA wsaData;
	SOCKET ListenSocket = INVALID_SOCKET;
    SOCKET ClientSocket = INVALID_SOCKET;

	struct addrinfo *result = NULL;
    struct addrinfo hints;

    std::vector<char> myRecv;
    
	char recvbuf[DEFAULT_BUFLEN];
    int recvbuflen = DEFAULT_BUFLEN;
    
    // Initialize Winsock
    iResult = WSAStartup(MAKEWORD(2,2), &wsaData);
    if (iResult != 0) {
        printf("WSAStartup failed with error: %d\n", iResult);
        return 1;
    }

    ZeroMemory(&hints, sizeof(hints));
    hints.ai_family = AF_INET;
    hints.ai_socktype = SOCK_STREAM;
    hints.ai_protocol = IPPROTO_TCP;
    hints.ai_flags = AI_PASSIVE;

    // Resolve the server address and port
    iResult = getaddrinfo(NULL, DEFAULT_PORT, &hints, &result);
    if ( iResult != 0 ) {
        printf("getaddrinfo failed with error: %d\n", iResult);
        WSACleanup();
        return 1;
    }

    // Create a SOCKET for connecting to server
    ListenSocket = socket(result->ai_family, result->ai_socktype, result->ai_protocol);
    if (ListenSocket == INVALID_SOCKET) {
        printf("socket failed with error: %ld\n", WSAGetLastError());
        freeaddrinfo(result);
        WSACleanup();
        return 1;
    }

    // Setup the TCP listening socket
    iResult = bind( ListenSocket, result->ai_addr, (int)result->ai_addrlen);
    if (iResult == SOCKET_ERROR) {
        printf("bind failed with error: %d\n", WSAGetLastError());
        freeaddrinfo(result);
        closesocket(ListenSocket);
        WSACleanup();
        return 1;
    }

    freeaddrinfo(result);

    iResult = listen(ListenSocket, SOMAXCONN);
    if (iResult == SOCKET_ERROR) {
        printf("listen failed with error: %d\n", WSAGetLastError());
        closesocket(ListenSocket);
        WSACleanup();
        return 1;
    }

    // Accept a client socket
    ClientSocket = accept(ListenSocket, NULL, NULL);
    if (ClientSocket == INVALID_SOCKET) {
        printf("accept failed with error: %d\n", WSAGetLastError());
        closesocket(ListenSocket);
        WSACleanup();
        return 1;
    }

    // No longer need server socket
    closesocket(ListenSocket);

    // Receive until the peer shuts down the connection
    do {

        iResult = recv(ClientSocket, recvbuf, recvbuflen, 0);
        if (iResult > 0) {
			
			std::vector<char> vec(recvbuf, recvbuf + iResult);
			std::string myRecv(vec.begin(), vec.end());

			// END                       - Terminates connection and program. 
			if (myRecv == "END\r\n") { 
	
			CyQu_EXIT(ClientSocket);

			}

			std::cout << myRecv;
		    char split_char = ' ';
            std::istringstream split(myRecv);
            std::vector<std::string> myCmd2;

            for (std::string each; std::getline(split, each, split_char); myCmd2.push_back(each));

		

			if (myCmd2.size() < 2) {
				
			 myRecv = "";
			 myCmd2.resize(0);
			 continue;
			} 

			// ADD     [TABLE] [DATA]    - Adds [DATA] to [TABLE] where [DATA] is in 1.1.1 format 
			if (myCmd2[0] == "ADD") { 
	
			CQ_success = CyQu_ADD(ClientSocket, myCmd2[1]);

			}

			// GET     [TABLE] [INDEX]   - Retrieves "1.2.3" string of data by [INDEX] starting with integer 1 in the order added.
	     	if (myCmd2[0] == "GET") { 
	
				CQ_success = CyQu_GET(ClientSocket, myCmd2[1]);

				if (CQ_success == 0) {
				 cout << "\n Out of Range \n"; 
				 CySend(ClientSocket, "CY: OUT OF RANGE");
				}

			}
			
			// UPDATE  [TABLE]           - Refreshes [TABLE] arrays in GPU memory, that was inserted by ADD.
			if (myCmd2[0] == "UPDATE") {
			 
			}

			if (myCmd2[0] == "FIND") {
		     CQ_success = CyQu_FIND(ClientSocket, myCmd2[1]);
			}
			
			if (myCmd2[0] == "CLEAR") {
			 CySend(ClientSocket, "Database Cleared");
		     CQ_success = CyQu_CLEAR(ClientSocket);
			}

			// Programmers Notes:
			// Using   [TABLE] will probably require use of a 2d array and &pointer setup to reference it.
			//          where [TABLE] is likely defined as an integer index. For now, we will stick to a
			//          flat array for proof of concept.

			// MAKE    [TABLE] 
			// ADD     [TABLE] [DATA]         - Adds [DATA] to [TABLE] where [DATA] is in 1.1.1 format 
            // GET     [TABLE] [INDEX]        - Retrieves "1.2.3" string of data by [INDEX] starting with integer 1 in the order added.
			// UPDATE  [TABLE]                - Refreshes [TABLE] arrays in GPU memory, that was inserted by ADD.
			// FIND    [TABLE] [DATA]         - Searches for [DATA] 1.1.1 which is converted to CPU array, and sent to GPU memory.
			// STOP    [TABLE] [SEARCH_INDEX] - Stops searching for [DATA] in the provided [SEARCH_INDEX] 
			// FREE    [TABLE]                - Frees up [TABLE] array from local memory and GPU memory.
			// ACTIVE                         - Outputs all active search indexes in form of 
			//                                   :ACTIVE_TOTAL [TOTAL_ACTIVE_SEARCHES] 
			//                                   :ACTIVE       [SEARCH_INDEX] [SEARCH_ARRAY_STRING] [TOTAL_FOUND_RESULTS]
			// 
			// SAVE    [TABLE] [FILE]    - Saves [TABLE] and outputs [TABLE] and [FILE] saved to.
			// LOAD    [TABLE] [FILE]    - Loads [TABLE] and outputs [TABLE] and [FILE] loaded from.
			// END                       - Terminates connection and program. 


        }
        else if (iResult == 0)
            printf("Connection closing...\n");
        else  {
            printf("recv failed with error: %d\n", WSAGetLastError());
            closesocket(ClientSocket);
            WSACleanup();
            return 1;
        }

    } while (iResult > 0);

    // shutdown the connection since we're done
    iResult = shutdown(ClientSocket, SD_SEND);
    if (iResult == SOCKET_ERROR) {
        printf("shutdown failed with error: %d\n", WSAGetLastError());
        closesocket(ClientSocket);
        WSACleanup();
        return 1;
    }

    // cleanup
    closesocket(ClientSocket);
    WSACleanup();

    return 0;
}


// Main Function.
int main(int argc, const char *argv[]) {
	
	init_Server();

	return 0;
	
}
