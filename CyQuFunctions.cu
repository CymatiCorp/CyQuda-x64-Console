#include "hip/hip_runtime.h"
#define WIN32_LEAN_AND_MEAN
#include <stdio.h>
#include <assert.h>
#include <string>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <iostream>
#include <vector>


#include "CyQu_bridge.h"

//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>

/*
 TO DO
  1. Search Entries using CUDA.
   2. Load entries from file
    3. Use a socket layer to accept commands.

  For "search entries", to copy the Array from Host to Device 
  we will need to flatten the array 
    from [Row][Column] 
	  to [Row + Column*N]
 
  Decide between Array, Vector, or Thrust
  where each has its drawbacks and limitations.

  * Do parallel searches of before.current.after words.
  * 128 messages per 8 word index 128*8=1024 , Y*X
   
  */

using namespace std;

static const signed int arraySpace = 64;
static const size_t mySize = arraySpace * sizeof(signed int);

//Initialize Variables.
int matrixIndex = -1;    // Initial matrix Index.
int searchIndex = 0;    // Initial search Index.

int matrixSize = 8000;  // Max argvbase Entries.
int searchSize = 10;    // Max Search Entries.
int resultSize = 20;    // Max Result Entries.

int selectIndex = 0;
int i = 0;

// Create Arrays.

signed int* myMatrix = (signed int*)malloc(mySize);
signed int *deviceMatrix;
vector<vector<int>> mySearch;
vector<int> myResults;

vector<vector<int>> *deviceSearch;
vector<vector<int>> *deviceResults;


__global__ void CyQu_Kernel(int* deviceMatrix) {

	// printf("Block XY [%d, %d] ", blockIdx.x, blockIdx.y);
	// printf("\n");
	// printf("Grid X [%d] ", gridDim.x);
	// printf("\n");
	 printf("Block XY [%d, %d] : Grid [%d] :  Thread XY [%d, %d] \n", blockIdx.x, blockIdx.y, gridDim.x, threadIdx.x, threadIdx.y);
	// printf("[%d][%d]\n",threadIdx.y,threadIdx.x);
	//if (threadIdx.y > matrixIndex) { return; }
	//int selector = threadIdx.x * threadIdx.y;
	for (int i = 0; i < 8; i++) {
    // printf("[%d] = [%d] \n",i, deviceMatrix[selector]);
	}
	// printf("\n");
	// printf("\n");
	/*
	int idx = threadIdx.x;
	int idy = threadIdx.y
	
	for (i = 0; i < 9; i++) {
       if (&deviceMatrix[idx][idy] == &deviceSearch[selectedIndex][i]) {
	    CySend(ClientSocket, "");
	   }

	}
*/     

}

int CyQu_CLEAR (SOCKET ClientSocket) {
	
    free(myMatrix);
	hipFree(deviceMatrix);

	signed int* myMatrix = (signed int*)malloc(mySize);
    signed int *deviceMatrix;

	matrixIndex = -1;

	cout << "Matrix Cleared \n";
	
	return 1;
}

int CyQu_USAGE (SOCKET ClientSocket, string cyData) {
		if (cyData == "ADD") { cout << "\n\n Usage: ADD 1.2.3 \n\n"; }
		if (cyData == "GET") { cout << "\n\n Usage: GET 1 \n\n"; }
		if (cyData == "Delete") { cout << "\n\n Usage: Delete 1 \n\n"; }
		if (cyData == "Search") { cout << "\n\n Usage: Search 1.2.3 \n\n"; }
return 1;
}

int CyQu_ADD (SOCKET ClientSocket, string mycyData) {

	char split_char2 = '.';
    std::istringstream split2(mycyData);
    std::vector<std::string> token;
    for (std::string each2; std::getline(split2, each2, split_char2); token.push_back(each2));
	
     matrixIndex++;
	 
	 if (sizeof(myMatrix) < 1) { 
		 // myMatrix.resize(myMatrix.size() + 1000);

      
	  
	 }
	 // myMatrix[matrixIndex].resize(token.size() + 1);
	
	 for (i = 0; i < token.size(); i++) {
      myMatrix[(matrixIndex * 8 + i)] = atoi(token[i].c_str());
	 }
	
return 1;
}

//int CyQu_SEARCH (SOCKET ClientSocket, string mycyData) {

	
//return 1
//}

int CyQu_GET (SOCKET ClientSocket, string cyData) {
	
        selectIndex = atoi(cyData.c_str());

		if (selectIndex > matrixIndex) { return 0;	}

		if (selectIndex < 0) { 	return 0; }

		string myResult = "CY: ";

  	    printf("[%d]", selectIndex);

		for (i = 0; i < 8; i++) {
   	     myResult = myResult + " " + to_string(myMatrix[selectIndex * 8 + i]); 
		} 

		CySend(ClientSocket, myResult);

  return 1;
}

int CyQu_FIND (SOCKET ClientSocket, string mycyData) {

	if (matrixIndex == 0) { 
		 cout << "- Database Empty.\n";
		 return 0;
	 }

	if (mySearch.size() < 1) { 
		mySearch.resize(10); 
	}

	char split_char2 = '.';
    std::istringstream split2(mycyData);
    std::vector<std::string> token;
    for (std::string each2; std::getline(split2, each2, split_char2); token.push_back(each2));

	searchIndex = atoi(token[0].c_str());

	if (searchIndex >= searchSize) { 
		 cout << "- Search Index Out of Range. Use 0- \n" << (searchSize -1); 
		 return 0; 
	 }


     mySearch[searchIndex].resize((token.size() -1));
	
	 for (i = 1; i < (token.size() -1); i++) {
      mySearch[searchIndex][(i - 1)] = atoi(token[i].c_str());
	 }
     
	 hipSetDevice(0);

	 //int deviceMatrixSize = sizeof(myMatrix) * sizeof(signed int);
	 hipMalloc((void**)&deviceMatrix, mySize);
     
	 /*
	 int deviceSearchSize = mySearch.size() * sizeof(int);
	 hipMalloc((void**)&deviceSearch, deviceSearchSize);
     hipMemcpy(deviceSearch, &mySearch[0], deviceSearchSize, hipMemcpyHostToDevice);
	 */

 	 //int deviceResultsSize = myResults.size() * sizeof(int);
	 //hipMalloc((void**)&deviceResults, deviceResultsSize);
     //hipMemcpy(deviceResults, &myResults[0], deviceResultsSize, hipMemcpyHostToDevice);

	 for (int i = 0; i < 16; i++) {
      printf("[%d] = [%d] \n",i, myMatrix[i]);
	 }

	 cout << "Kernal>>> \n\n";
	 
	 dim3 dimGrid(1);
	 dim3 dimBlock(8, matrixIndex +1);

     hipMemcpy(deviceMatrix, &myMatrix[0], mySize, hipMemcpyHostToDevice);

	 CyQu_Kernel<<<dimGrid, dimBlock>>>(deviceMatrix);

	 return 1;
}



void CyQu_LOAD(SOCKET ClientSocket, string cyLoad) {

}

void CyQu_SAVE(SOCKET ClientSocket, string cySave) {

}

void CyQu_EXIT(SOCKET ClientSocket) {
        free(myMatrix);
	    hipFree(deviceMatrix);

	    cout << "\n\n Exiting . . .\n\n";
		closesocket(ClientSocket);
        WSACleanup();
		exit(EXIT_WAIVED);
return;

}
